#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hip/hip_vector_types.h>
#include <iostream>

// Problem 1: Kernel to add two vectors
__global__ void addVectors(float *a, float *b, float *c, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        c[id] = a[id] + b[id];
    }
}

// Kernel to scale random numbers to [-1, 1]
__global__ void scaleKernel(float *data, size_t size, float scale, float shift)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        data[id] = data[id] * scale + shift;
    }
}

void problem1_cuda()
{
    const size_t size = 1 << 24; // 2^24
    float *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_c, size * sizeof(float));

    // Initialize CURAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Generate random numbers in [0, 1) and scale to [-1, 1]
    // CURAND generates random floats between 0 and 1. We then scale the values to be between -1 and 1.
    hiprandGenerateUniform(gen, d_a, size);
    hiprandGenerateUniform(gen, d_b, size);

    // Scale to [-1, 1]
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, size, 2.0f, -1.0f);
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_b, size, 2.0f, -1.0f);

    // Add vectors
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy the first element of c back to host
    float h_c;
    hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Problem 1: First element of c: " << h_c << std::endl;

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hiprandDestroyGenerator(gen);
}

// Problem 2: Kernel to normalize 4D vectors
__global__ void normalizeVectors(float4 *vec, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        float4 v = vec[id];
        float norm = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z + v.w * v.w);
        if (norm > 0.0f)
        {
            vec[id].x /= norm;
            vec[id].y /= norm;
            vec[id].z /= norm;
            vec[id].w /= norm;
        }
    }
}

void problem2_cuda()
{
    const size_t size = 1 << 22; // 2^22
    float4 *d_vec;

    // Allocate device memory
    hipMalloc(&d_vec, size * sizeof(float4));

    // Initialize CURAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Generate random numbers into a flat float array
    float *d_data;
    hipMalloc(&d_data, 4 * size * sizeof(float));
    hiprandGenerateUniform(gen, d_data, 4 * size);

    // Scale to [-1, 1]
    int threadsPerBlock = 256;
    int blocksPerGrid = (4 * size + threadsPerBlock - 1) / threadsPerBlock;
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, 4 * size, 2.0f, -1.0f);

    // Copy to float4 array
    hipMemcpy(d_vec, d_data, size * sizeof(float4), hipMemcpyDeviceToDevice);
    hipFree(d_data);

    // Normalize the vectors
    // Each thread computes the norm of a 4D vector and normalizes it. This is a parallel operation that benefits from the GPU’s many cores.
    blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    normalizeVectors<<<blocksPerGrid, threadsPerBlock>>>(d_vec, size);

    // Copy the first normalized vector back to host
    float4 h_vec;
    hipMemcpy(&h_vec, d_vec, sizeof(float4), hipMemcpyDeviceToHost);
    std::cout << "Problem 2: First normalized vector: " << h_vec.x << ", " << h_vec.y << ", " << h_vec.z << ", " << h_vec.w << std::endl;

    // Clean up
    hipFree(d_vec);
    hiprandDestroyGenerator(gen);
}
void measureCuda(void (*func)(), const std::string &label)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    func();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << label << " execution time: " << ms << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main()
{
    problem1_cuda();
    problem2_cuda();
    measureCuda(problem1_cuda, "Problem 1");
    measureCuda(problem2_cuda, "Problem 2");
    return 0;
}
/*
 * Problem 1 execution time: 3.9095 ms
 * Problem 2 execution time: 3.15635 ms
 *  The execution time on the GPU is in the millisecond range for both problems, showcasing the speed-up when using a GPU compared to sequential CPU execution.
 * The GPU handles the problem efficiently by utilizing thousands of threads to compute the vector operations concurrently.

 */