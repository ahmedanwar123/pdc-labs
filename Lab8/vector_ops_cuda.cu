#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hip/hip_vector_types.h>
#include <iostream>

// Problem 1: Kernel to add two vectors
__global__ void addVectors(float *a, float *b, float *c, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        c[id] = a[id] + b[id];
    }
}

// Kernel to scale random numbers to [-1, 1]
__global__ void scaleKernel(float *data, size_t size, float scale, float shift)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        data[id] = data[id] * scale + shift;
    }
}

void problem1_cuda()
{
    const size_t size = 1 << 24; // 2^24
    float *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc(&d_a, size * sizeof(float));
    hipMalloc(&d_b, size * sizeof(float));
    hipMalloc(&d_c, size * sizeof(float));

    // Initialize CURAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Generate random numbers in [0, 1) and scale to [-1, 1]
    hiprandGenerateUniform(gen, d_a, size);
    hiprandGenerateUniform(gen, d_b, size);

    // Scale to [-1, 1]
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, size, 2.0f, -1.0f);
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_b, size, 2.0f, -1.0f);

    // Add vectors
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy the first element of c back to host
    float h_c;
    hipMemcpy(&h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Problem 1 (CUDA): First element of c: " << h_c << std::endl;

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hiprandDestroyGenerator(gen);
}

// Problem 2: Kernel to normalize 4D vectors
__global__ void normalizeVectors(float4 *vec, size_t size)
{
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
    {
        float4 v = vec[id];
        float norm = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z + v.w * v.w);
        if (norm > 0.0f)
        {
            vec[id].x /= norm;
            vec[id].y /= norm;
            vec[id].z /= norm;
            vec[id].w /= norm;
        }
    }
}

void problem2_cuda()
{
    const size_t size = 1 << 22; // 2^22
    float4 *d_vec;

    // Allocate device memory
    hipMalloc(&d_vec, size * sizeof(float4));

    // Initialize CURAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    // Generate random numbers into a flat float array
    float *d_data;
    hipMalloc(&d_data, 4 * size * sizeof(float));
    hiprandGenerateUniform(gen, d_data, 4 * size);

    // Scale to [-1, 1]
    int threadsPerBlock = 256;
    int blocksPerGrid = (4 * size + threadsPerBlock - 1) / threadsPerBlock;
    scaleKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, 4 * size, 2.0f, -1.0f);

    // Copy to float4 array
    hipMemcpy(d_vec, d_data, size * sizeof(float4), hipMemcpyDeviceToDevice);
    hipFree(d_data);

    // Normalize the vectors
    blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    normalizeVectors<<<blocksPerGrid, threadsPerBlock>>>(d_vec, size);

    // Copy the first normalized vector back to host
    float4 h_vec;
    hipMemcpy(&h_vec, d_vec, sizeof(float4), hipMemcpyDeviceToHost);
    std::cout << "Problem 2 (CUDA): First normalized vector: " << h_vec.x << ", " << h_vec.y << ", " << h_vec.z << ", " << h_vec.w << std::endl;

    // Clean up
    hipFree(d_vec);
    hiprandDestroyGenerator(gen);
}

int main()
{
    problem1_cuda();
    problem2_cuda();
    return 0;
}